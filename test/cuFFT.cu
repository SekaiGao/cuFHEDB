#include "hip/hip_runtime.h"
#include "HEDB/comparison/comparison.h"
#include "HEDB/comparison/tfhepp_utils.h"
#include "HEDB/conversion/repack.h"
#include "HEDB/utils/types.h"
#include "HEDB/utils/utils.h"
#include "cuHEDB/HomCompare_gpu.cuh"
//#include "fastR.h"
#include <chrono>
#include <hipfft/hipfft.h>
#include <iomanip>
#include <omp.h>
#include <random>
#include <unistd.h>

using namespace HEDB;
using namespace seal;

template<class P>
void generateData(std::array<typename P::T, P::n> &trlwe) {
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_int_distribution<typename P::T> dis(0, 18446744073700);

  for (int i = 0; i < P::n; i++) {
    uint32_t value = dis(gen);
    trlwe[i] = value;
  }
}

template<class P>
void computeIFFT(const std::array<typename P::T, P::n>& input, std::array<double, P::n>& output,uint32_t test_num, double&cost) {
    std::vector<double> input_double(P::n);  
    for (int i = 0; i < P::n; i++) {
        input_double[i] = static_cast<double>(input[i]);
    }

    std::chrono::system_clock::time_point start, end;

    hipfftDoubleComplex* d_input;
    hipfftDoubleComplex* d_output;
    hipMalloc(&d_input, sizeof(hipfftDoubleComplex) * P::n);
    hipMalloc(&d_output, sizeof(hipfftDoubleComplex) * P::n);
    
    hipMemcpy(d_input, input_double.data(), sizeof(hipfftDoubleComplex) * P::n, hipMemcpyHostToDevice);


    hipfftHandle plan;

	  start = std::chrono::system_clock::now();

	  for(int i=0;i<test_num;++i) {
	  	hipfftPlan1d(&plan, P::n, HIPFFT_Z2Z, 1);
      hipfftExecZ2Z(plan, d_input, d_output, HIPFFT_BACKWARD);
	  	hipDeviceSynchronize();
	  }

	  end = std::chrono::system_clock::now();
    cost = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();


    std::vector<hipfftDoubleComplex> host_output(P::n);
    hipMemcpy(host_output.data(), d_output, sizeof(hipfftDoubleComplex) * P::n, hipMemcpyDeviceToHost);

    for (int i = 0; i < P::n; i++) {
        output[i] = host_output[i].x / P::n; 
    }

    hipfftDestroy(plan);
    hipFree(d_input);
    hipFree(d_output);
}

__global__ void emptyKernel() {
}

int main()
{
  std::cout<<"1\n";
  #if 0
  uint32_t test_num = 1000;
  omp_set_num_threads(num_stream1);
  warmupGPU();

  TFHEpp::Polynomial<Lvl1> tlwefft;
  std::array<double, Lvl1::n> fftb1, fftb2;

  generateData<Lvl1>(tlwefft);

  double costs;
  std::chrono::system_clock::time_point start, end;

  start = std::chrono::system_clock::now();
  costs = 0;
  for (int i = 0; i < test_num; ++i) {
    emptyKernel<<<1, 64>>>();
    hipDeviceSynchronize();
  }
  end = std::chrono::system_clock::now();
  costs = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

  std::cout << "kernel launch overhead: " << costs / test_num << "μs." << std::endl;

  
  cufftlvl1.ifft_test(fftb2, tlwefft, test_num, costs);

  std::cout << "negacyclic fft gpu: " << costs / test_num << "μs." << std::endl;
  computeIFFT<Lvl1>(tlwefft, fftb1, test_num, costs);
  std::cout << "cufft: " << costs / test_num << "μs." << std::endl;

  start = std::chrono::system_clock::now();
  costs = 0;
  for (int i = 0; i < test_num; ++i) {
    TFHEpp::TwistIFFT<Lvl1>(fftb1, tlwefft);
  }
  end = std::chrono::system_clock::now();
  costs = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
  std::cout << "negacyclic fft cpu: " << costs / test_num << "μs." << std::endl;


  double err =0;
  for(int i=0;i<Lvl1::n;++i) {
	err+=fabs(fftb1[i]-fftb2[i]);
  }
  std::cout<<"error: "<<err<<std::endl;
  #endif
  return 0;
}
